#include "hip/hip_runtime.h"
/**
 * Program Diversity index computes an approximation of pi in parallel on the GPU by
 * generating N random (x,y) points in the unit square and counting how many
 * fall within a distance of 1 from the origin.
 *
 * Usage: PiGPU <seed> <trials> <pop1> <pop2> ...
 * <seed> = Pseudorandom number generator seed
 * <trials> = Number of trials, trials >= 1
 * <pop1> = population in group 1, pop1 >= 1 
 */

#include <stdlib.h>
#include <stdio.h>
#include <hip/hip_runtime_api.h>

#include "Util.cu"
#include "Random.cu"

//------------------------------------------------------------------------------
// DEVICE FUNCTIONS

// Number of threads per block.
#define NT 1024

// Overall counter variable in global memory.
__device__ unsigned long long int devCount;

// Per-thread counter variables in shared memory.
__shared__ unsigned long long int shrCount [NT];

/**
 * Device kernel to compute Diversity index.
 *
 * Called with a one-dimensional grid of one-dimensional blocks, NB blocks, NT
 * threads per block.
 *
 * @param  seed  Pseudorandom number generator seed.
 * @param  trials     Number of trials.
 * @param  numberOfArgs  Number of populations   
 * @param  total      total number of people
 * @param  dev_sectionsSP   Starting range for all sections
 * @param  dev_sectionsEP   Ending range for all sections
 */
__global__ void computeDiversityIndex
	(unsigned long long int seed, 
	 unsigned long long int trials, 
	 int numberOfArgs, 
	 int total,
	 int *dev_sectionsSP,
	 int *dev_sectionsEP){
   
   // Declare variables in kernel
   int x, size, rank;
   int catigory1 = 0;
   int catigory2 = 0;
   unsigned long long int thrTrialsSize, lb, ub, count;
   int random_person1 = 0;
   int random_person2 = 0;

   // Pseudo random number generator
   prng_t prng;

   // Determine number of threads and this thread's rank.
   x = threadIdx.x;
   size = gridDim.x*NT;
   rank = blockIdx.x*NT + x;

   // Determine iterations for this thread.
   thrTrialsSize = (trials + size - 1)/size;
   lb = rank*thrTrialsSize;
   ub = min (lb + thrTrialsSize, trials) - 1;

   // Initialize per-thread prng and count.
   prngSetSeed (&prng, seed + rank);
   count = 0;

	  // Pick random people and test if they are in the same population
	  for(int t = lb; t <= ub; t++){

		random_person1 = prngNextInt(&prng, total )+ 1 ;
		random_person2 = prngNextInt(&prng, total )+ 1 ;

		while(random_person2 == random_person1){
			random_person1 = prngNextInt(&prng, total )+ 1 ;
			random_person2 = prngNextInt(&prng, total )+ 1 ;
		}

		for(int i = 0; i < numberOfArgs; i++){
			
			if(random_person1 >= dev_sectionsSP[i] && random_person1 <= dev_sectionsEP[i]){
				catigory1 = i;
			}
			
			if(random_person2 >= dev_sectionsSP[i] && random_person2 <= dev_sectionsEP[i]){
				catigory2 = i;
			}
		}

		if(catigory1 != catigory2){
			count++;
		}

		catigory1 = 0;
		catigory2 = 0;
	  }
	  

   // Shared memory parallel reduction within thread block.
   shrCount[x] = count;
   __syncthreads();
   for (int i = NT/2; i > 0; i >>= 1){
      if (x < i){
         shrCount[x] += shrCount[x+i];
		 }
      __syncthreads();
      }

   // Atomic reduction into overall counter.
   if (x == 0)
      atomicAdd (&devCount, shrCount[0]);
   }

//------------------------------------------------------------------------------
// HOST FUNCTIONS

/**
 * Print a usage message and exit.
 */
static void usage()
   {
   fprintf (stderr, "Usage: DiversityIndex <seed> <trials> <pop1> <pop2> ...\n");
   fprintf (stderr, "<seed> = Pseudorandom number generator seed\n");
   fprintf (stderr, "<trials> = Number of trials, N >= 1\n");
   fprintf (stderr, "<pop> = population count, pop >= 1\n");
   exit (1);
   }

/**
 * Main program.
 */
int main(int argc, char *argv[]){

   // declare variables
   unsigned long long int seed, trials, hostCount;
   int dev, NB;
   int numberOfArgs = 0;
   
   // Minus 3 because 0 is the program name, 1 is the seed, 2 is the trials
   numberOfArgs = argc - 3;
   int arguments[numberOfArgs];
   int temp;
   
   int sectionsSP[numberOfArgs];
   int *dev_sectionsSP;
   size_t SPbytes = numberOfArgs * sizeof(int);
   
   
   int sectionsEP[numberOfArgs];
   int *dev_sectionsEP;
   size_t EPbytes = numberOfArgs * sizeof(int);
   
   int total = 0;
   
   
   // Parse command line arguments.
   if (argc < 4) usage();
   progname = argv[0];
   if (sscanf (argv[1], "%llu", &seed) != 1) usage();
   if (sscanf (argv[2], "%llu", &trials) != 1 || trials < 1) usage();
   
    for(int i = 0; i < numberOfArgs; i++){
		if (sscanf (argv[i+3], "%d", &temp) != 1) usage();
		arguments[i] = temp;
    }
	
	// If one <pop> and 1 person in that population print and exit
	if(numberOfArgs == 1 && arguments[0] == 1){
		printf ("Diversity index = 0/0 = 0.0\n");
		return 0;
	}
   
   
   
    // calculate the ranges for each population
    sectionsSP[0] = 1;
	sectionsEP[0] = arguments[0];
	total = arguments[0];
	
   	for(int i = 1; i < numberOfArgs; i++ ){
		sectionsSP[i] = sectionsEP[i -1] + 1;
		total = sectionsEP[i] = (sectionsEP[i- 1] ) + arguments[i];
	}
	

   // Set CUDA device and determine number of multiprocessors (thread blocks).
   dev = setCudaDevice();
   checkCuda
      (hipDeviceGetAttribute (&NB, hipDeviceAttributeMultiprocessorCount, dev),
       "Cannot get number of multiprocessors");

   // Initialize overall counter.
   hostCount = 0;
   
    // allocate memory on card for dev_sections arrays
    checkCuda (hipMalloc (&dev_sectionsSP, SPbytes), 
	  "Cannot allocate dev_sectionsSP");
	  
	checkCuda (hipMalloc (&dev_sectionsEP, EPbytes),
      "Cannot allocate dev_sectionsEP");
   
	// Copy sections SP and EP arrays to device.
    checkCuda (hipMemcpy (dev_sectionsSP, sectionsSP, SPbytes, hipMemcpyHostToDevice),
      "Cannot upload dev_sectionsSP");
	  
    checkCuda (hipMemcpy (dev_sectionsEP, sectionsEP, EPbytes, hipMemcpyHostToDevice),
      "Cannot upload dev_sectionsEP");
   
	// Copy host count to device count 
    checkCuda(hipMemcpyToSymbol(HIP_SYMBOL(devCount), &hostCount, sizeof(hostCount)),
      "Cannot initialize devCount");

	   
   // Compute Diversity index in parallel on the GPU. 
   computeDiversityIndex <<< NB, NT >>> (seed, trials, numberOfArgs, total, dev_sectionsSP, dev_sectionsEP);
   hipDeviceSynchronize();
   checkCuda
      (hipGetLastError(),
       "Cannot launch computeDiversityIndex() kernel");

   // Get overall counter from GPU.
   checkCuda
      (hipMemcpyFromSymbol(&hostCount, HIP_SYMBOL(devCount), sizeof(hostCount)),
       "Cannot copy devCount to hostCount");
	
	
   // Print results.
   printf ("Diversity index = %llu/%llu = %.1f\n", hostCount, trials, 100.0*hostCount/trials);
   }
